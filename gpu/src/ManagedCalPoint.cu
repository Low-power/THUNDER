#include "Config.h"

#include "ManagedCalPoint.h"
#include "Device.cuh"

void ManagedCalPoint::Init(int mode, int cSearch, int gpuIdx, int nR, int nT, int mD, int npxl)
{
    _mode = mode;
    _cSearch = cSearch;
    _deviceId = gpuIdx;
    _nR = nR;
    _nT = nT;
    _mD = mD;
    
    hipSetDevice(gpuIdx); 

    /* Create and setup cuda stream */
    stream = (hipStream_t*)malloc(sizeof(hipStream_t));

    hipStreamCreate((hipStream_t*)stream);
    cudaCheckErrors("Create Stream.");
    
    hipMalloc((void**)&priRotP, nR * npxl * sizeof(Complex));
    cudaCheckErrors("Allocate rotP data.");

    hipMalloc((void**)&devtraP, nT * npxl * sizeof(Complex));
    cudaCheckErrors("Allocate traP data.");
    
    if (cSearch != 2)
    {
        hipMalloc((void**)&devDvp, nR * nT * sizeof(RFLOAT));
        cudaCheckErrors("Allocate dvP data.");
    
        hipMalloc((void**)&devD, sizeof(double));
        cudaCheckErrors("Allocate d data.");
    }
    else
    {
        hipMalloc((void**)&devctfD, mD * npxl * sizeof(RFLOAT));
        cudaCheckErrors("Allocate ctfP data.");
        
        hipMalloc((void**)&devdP, mD * sizeof(double));
        cudaCheckErrors("Allocate frequence data.");
        
        hipMalloc((void**)&devDvp, nR * nT * mD * sizeof(RFLOAT));
        cudaCheckErrors("Allocate dvP data.");
        
        hipMalloc((void**)&devtT, nR * nT * sizeof(RFLOAT));
        cudaCheckErrors("Allocate tT data.");
    
        hipMalloc((void**)&devtD, nR * mD * sizeof(RFLOAT));
        cudaCheckErrors("Allocate tD data.");
    
        hipMalloc((void**)&devD, mD * sizeof(double));
        cudaCheckErrors("Allocate d data.");
    } 
    
    hipMalloc((void**)&devBaseL, sizeof(RFLOAT));
    cudaCheckErrors("Allocate w data.");
    
    hipMalloc((void**)&devwC, sizeof(RFLOAT));
    cudaCheckErrors("Allocate wc data.");
    
    hipMalloc((void**)&devwR, nR * sizeof(RFLOAT));
    cudaCheckErrors("Allocate wr data.");
    
    hipMalloc((void**)&devwT, nT * sizeof(RFLOAT));
    cudaCheckErrors("Allocate wt data.");
    
    hipMalloc((void**)&devwD, mD * sizeof(RFLOAT));
    cudaCheckErrors("Allocate wd data.");
    
    hipMalloc((void**)&devR, nR * sizeof(double));
    cudaCheckErrors("Allocate r data.");
    
    hipMalloc((void**)&devT, nT * sizeof(double));
    cudaCheckErrors("Allocate t data.");
    
    hipMalloc((void**)&devnR, nR * 4 * sizeof(double));
    cudaCheckErrors("Allocate nR data.");
    
    if (mode == 1)
    {
        hipMalloc((void**)&devnR, nR * 4 * sizeof(double));
        cudaCheckErrors("Allocate nR data.");
    
        hipMalloc((void**)&devRotm, nR * 9 * sizeof(double));
        cudaCheckErrors("Allocate rotM data.");
    }
    else
    {
        hipMalloc((void**)&devnR, nR * 2 * sizeof(double));
        cudaCheckErrors("Allocate nR data.");
    }

    hipMalloc((void**)&devnT, nT * 2 * sizeof(double));
    cudaCheckErrors("Allocate nT data.");
}

ManagedCalPoint::~ManagedCalPoint()
{
	// do clean up
    hipSetDevice(_deviceId);
    
    hipFree(priRotP);
    hipFree(devtraP);
    if (_cSearch == 2)
    {
        hipFree(devctfD);
        hipFree(devdP);
        hipFree(devtT);
        hipFree(devtD);
    }
    hipFree(devBaseL);
    hipFree(devDvp);
    hipFree(devwC);
    hipFree(devwR);
    hipFree(devwT);
    hipFree(devwD);
    hipFree(devR);
    hipFree(devT);
    hipFree(devD);
    hipFree(devnR);
    hipFree(devnT);
    if (_mode == 1)
    {
        hipFree(devRotm);
    }

    hipStreamDestroy(*(hipStream_t*)stream);
    cudaCheckErrors("Calpoint free error.");
}

Complex* ManagedCalPoint::getPriRotP()
{
	return priRotP;
}

Complex* ManagedCalPoint::getDevtraP()
{
	return devtraP;
}

RFLOAT* ManagedCalPoint::getDevctfD()
{
	return devctfD;
}

RFLOAT* ManagedCalPoint::getDevBaseL()
{
	return devBaseL;
}

RFLOAT* ManagedCalPoint::getDevDvp()
{
	return devDvp;
}

RFLOAT* ManagedCalPoint::getDevwC()
{
	return devwC;
}

RFLOAT* ManagedCalPoint::getDevwR()
{
	return devwR;
}

RFLOAT* ManagedCalPoint::getDevwT()
{
	return devwT;
}

RFLOAT* ManagedCalPoint::getDevwD()
{
	return devwD;
}

double* ManagedCalPoint::getDevR()
{
	return devR;
}

double* ManagedCalPoint::getDevT()
{
	return devT;
}

double* ManagedCalPoint::getDevD()
{
	return devD;
}

RFLOAT* ManagedCalPoint::getDevtT()
{
	return devtT;
}

RFLOAT* ManagedCalPoint::getDevtD()
{
	return devtD;
}

double* ManagedCalPoint::getDevdP()
{
	return devdP;
}

double* ManagedCalPoint::getDevnR()
{
	return devnR;
}

double* ManagedCalPoint::getDevnT()
{
	return devnT;
}

double* ManagedCalPoint::getDevRotm()
{
	return devRotm;
}

void* ManagedCalPoint::getStream()
{
	return stream;
}

int ManagedCalPoint::getMode()
{
    return _mode;
}

int ManagedCalPoint::getCSearch()
{
    return _cSearch;
}

int ManagedCalPoint::getDeviceId()
{
    return _deviceId;    
}

int ManagedCalPoint::getNR()
{
    return _nR;
}

int ManagedCalPoint::getNT()
{
    return _nT;    
}

int ManagedCalPoint::getMD()
{
    return _mD;    
}
