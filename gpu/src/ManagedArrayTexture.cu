#include "Config.h"

#include "ManagedArrayTexture.h"
#include "Device.cuh"

class ManagedArrayTexture::DeviceStruct {
public:
    ~DeviceStruct();
	void Initialize2D(int vdim);
	void Initialize3D(int vdim);

public:
    int deviceId;
	hipChannelFormatDesc channelDesc;
	hipArray* symArray;
	struct hipResourceDesc resDesc;
	hipTextureDesc td;
	hipTextureObject_t texObject;
};

ManagedArrayTexture::DeviceStruct::~DeviceStruct()
{
    hipSetDevice(deviceId);
    hipDestroyTextureObject(texObject);
    hipFreeArray(symArray);
    cudaCheckErrors("symArray free error.");
}

void ManagedArrayTexture::DeviceStruct::Initialize2D(int vdim)
{
    hipSetDevice(deviceId);
    cudaCheckErrors("Set deviceID error.");

#ifdef SINGLE_PRECISION
    channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
#else
    channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindSigned);
#endif    

    hipMallocArray(&symArray, &channelDesc, vdim / 2 + 1, vdim);
    cudaCheckErrors("Allocate symArray data.");

    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = symArray;
    
    memset(&td, 0, sizeof(td));
    td.normalizedCoords = 0;
    td.addressMode[0] = hipAddressModeClamp;
    td.addressMode[1] = hipAddressModeClamp;
    td.readMode = hipReadModeElementType;

    hipCreateTextureObject(&texObject, &resDesc, &td, NULL);
    cudaCheckErrors("Create symArray texObject.");
}

void ManagedArrayTexture::DeviceStruct::Initialize3D(int vdim)
{
    hipSetDevice(deviceId);
    cudaCheckErrors("cuda Set Device.");

#ifdef SINGLE_PRECISION
    channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);
#else
    channelDesc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindSigned);
#endif    

    hipExtent extent;
    extent = make_hipExtent(vdim / 2 + 1, vdim, vdim);
    hipMalloc3DArray(&symArray, &channelDesc, extent);
    cudaCheckErrors("Allocate symArray data.");

    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = symArray;
    
    memset(&td, 0, sizeof(td));
    td.normalizedCoords = 0;
    td.addressMode[0] = hipAddressModeClamp;
    td.addressMode[1] = hipAddressModeClamp;
    td.addressMode[2] = hipAddressModeClamp;
    td.readMode = hipReadModeElementType;

    hipCreateTextureObject(&texObject, &resDesc, &td, NULL);
    cudaCheckErrors("Create symArray texObject.");
}

void ManagedArrayTexture::Init(int mode, int vdim, int gpuIdx)
{
	_cuda = new DeviceStruct();
    _cuda->deviceId = gpuIdx;
    if (mode == 1)
	    _cuda->Initialize3D(vdim);
    else
        _cuda->Initialize2D(vdim);
}

ManagedArrayTexture::~ManagedArrayTexture()
{
	// do clean up
    delete _cuda;
}

void* ManagedArrayTexture::GetArray()
{
	return static_cast<void*>(_cuda->symArray);
}

void* ManagedArrayTexture::GetTextureObject()
{
	return static_cast<void*>(&_cuda->texObject);
}

int ManagedArrayTexture::getDeviceId()
{
    return _cuda->deviceId;    
}

